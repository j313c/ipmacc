
#include <hip/hip_runtime.h>


/* DISCLAIMER: THIS CODE IS INCLUDED FROM CUDA SDK AND MODIFIED A BIT
 * TO BE USED IN THIS PROJECT
 */

//#define DATATYPE DiffIndicatorCompressed
//#define DATATYPE float

//static inline __device__ uint compareFunction1(const DATATYPE &right, const DATATYPE &left)
template<class DATATYPE>
static inline __device__ uint compareFunction1(const DATATYPE &left, const DATATYPE &right)
{
    //if(__isnan(left.diffIndicator1)){
    //    return false;
    //}else if(__isnan(right.diffIndicator1)){
    //    return true;
    //}else{
    //    return (left.diffIndicator1<right.diffIndicator1);
    //}
    if(left<right){
        return true;
    }else{
        return false;
    }
}
//static inline __device__ uint compareFunction2(const DATATYPE &right, const DATATYPE &left)
template<class DATATYPE>
static inline __device__ uint compareFunction2(const DATATYPE &left, const DATATYPE &right)
{
    //if(__isnan(left.diffIndicator1)){
    //    return false;
    //}else if(__isnan(right.diffIndicator1)){
    //    return true;
    //}else{
    //    return (left.diffIndicator1<=right.diffIndicator1);
    //}
    if(left<=right){
        return true;
    }else{
        return false;
    }
}


template<class DATATYPE>
inline __device__ uint binarySearchExclusive(DATATYPE val, DATATYPE *data, uint L, uint stride)
{
    if (L == 0)
    {
        return 0;
    }

    uint pos = 0;

    for (; stride > 0; stride >>= 1)
    {
        uint newPos = umin(pos + stride, L);

        if (compareFunction1<DATATYPE>(data[newPos-1],val))
        //if ((sortDir && (data[newPos - 1] < val)) || (!sortDir && (data[newPos - 1] > val)))
        {
            pos = newPos;
        }
    }

    return pos;
}

template<class DATATYPE>
inline __device__ uint binarySearchInclusive(DATATYPE val, DATATYPE *data, uint L, uint stride)
{
    if (L == 0)
    {
        return 0;
    }

    uint pos = 0;

    for (; stride > 0; stride >>= 1)
    {
        uint newPos = umin(pos + stride, L);

        if (compareFunction2<DATATYPE>(data[newPos-1],val))
        //if ((sortDir && (data[newPos - 1] <= val)) || (!sortDir && (data[newPos - 1] >= val)))
        {
            pos = newPos;
        }
    }

    return pos;
}

//template<class DATATYPE, class sort_dir>
//#define DATATYPE float
template<class DATATYPE>
__device__ void mergeSortSharedCall(
    DATATYPE *s_key,
    //DATATYPE *s_val,
    uint arrayLength,
    uint (*compare1)(const DATATYPE &, const DATATYPE &),
    uint (*compare2)(const DATATYPE &, const DATATYPE &)
)
{
    for (uint stride = 1; stride < arrayLength; stride <<= 1)
    {
        uint     lPos = threadIdx.x & (stride - 1);
        DATATYPE *baseKey = s_key + 2 * (threadIdx.x - lPos);
        //DATATYPE *baseVal = s_val + 2 * (threadIdx.x - lPos);

        __syncthreads();
        DATATYPE keyA = baseKey[lPos +      0];
        DATATYPE keyB = baseKey[lPos + stride];
        //DATATYPE valA = baseVal[lPos +      0];
        //DATATYPE valB = baseVal[lPos + stride];
        uint posA = binarySearchExclusive<DATATYPE>(keyA, baseKey + stride, stride, stride) + lPos;
        uint posB = binarySearchInclusive<DATATYPE>(keyB, baseKey +      0, stride, stride) + lPos;

        __syncthreads();
        baseKey[posA] = keyA;
        baseKey[posB] = keyB;
        //baseVal[posA] = valA;
        //baseVal[posB] = valB;
    }

    __syncthreads();
}

