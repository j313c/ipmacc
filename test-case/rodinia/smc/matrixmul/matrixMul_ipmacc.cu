#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <openacc.h>
#define IPMACC_MAX1(A)   (A)
#define IPMACC_MAX2(A,B) (A>B?A:B)
#define IPMACC_MAX3(A,B,C) (A>B?(A>C?A:(B>C?B:C)):(B>C?C:B))
#ifdef __cplusplus
#include "openacc_container.h"
#endif

#include <hip/hip_runtime.h>

#include <malloc.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>

#include <math.h>

#define LEN 1024
#define SIZE LEN * LEN

#define TYPE double
#define MIN(a, b)    (a < b ? a : b)


  __global__ void __generated_kernel_region_0(TYPE * a,TYPE * c,TYPE * b);
 
int main(int argc, char *argv[])
{
  int i;
#ifdef __NVCUDA__
  acc_init(acc_device_nvcuda);
#endif
#ifdef __NVOPENCL__
  acc_init(acc_device_nvocl);
#endif

  TYPE *a, *b, *c;
  
  TYPE *seq;
  
  a = (TYPE*)malloc(SIZE * sizeof(TYPE));
  b = (TYPE*)malloc(SIZE * sizeof(TYPE));
  c = (TYPE*)malloc(SIZE * sizeof(TYPE));
  seq = (TYPE*)malloc(SIZE * sizeof(TYPE));

  
  for (i = 0; i < SIZE; ++i) {
    
    a [i] = (TYPE)i;
    b [i] = (TYPE)2 * i;
    c [i] = 0.0f;
  }  

  unsigned long long int tic, toc;
  
  int k, j, l;
  for (k = 0; k < 3; k++) {
    printf("Calculation on GPU ... ");
    tic = clock();



	ipmacc_prompt((char*)"IPMACC: memory allocation c\n");
acc_present_or_create((void*)c,(SIZE+0)*sizeof(TYPE ));
ipmacc_prompt((char*)"IPMACC: memory allocation a\n");
acc_present_or_create((void*)a,(SIZE+0)*sizeof(TYPE ));
ipmacc_prompt((char*)"IPMACC: memory allocation b\n");
acc_present_or_create((void*)b,(SIZE+0)*sizeof(TYPE ));
	ipmacc_prompt((char*)"IPMACC: memory copyin c\n");
acc_pcopyin((void*)c,(SIZE+0)*sizeof(TYPE ));
ipmacc_prompt((char*)"IPMACC: memory copyin a\n");
acc_pcopyin((void*)a,(SIZE+0)*sizeof(TYPE ));
ipmacc_prompt((char*)"IPMACC: memory copyin b\n");
acc_pcopyin((void*)b,(SIZE+0)*sizeof(TYPE ));


{


    {



/* kernel call statement [0, -1]*/
{
dim3 __ipmacc_gridDim(1,1,1);
dim3 __ipmacc_blockDim(1,1,1);
__ipmacc_blockDim.x=16;
__ipmacc_gridDim.x=(((abs((int)((LEN))-(0+0)))/(1))/__ipmacc_blockDim.x)+(((((abs((int)((LEN))-(0+0)))/(1))%(16))==0?0:1));
__ipmacc_blockDim.y=16;
__ipmacc_gridDim.y=(((abs((int)((LEN))-(0+0)))/(1))/__ipmacc_blockDim.y)+(((((abs((int)((LEN))-(0+0)))/(1))%(16))==0?0:1));
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Launching kernel 0 > gridDim: (%u,%u,%u)\tblockDim: (%u,%u,%u)\n",__ipmacc_gridDim.x,__ipmacc_gridDim.y,__ipmacc_gridDim.z,__ipmacc_blockDim.x,__ipmacc_blockDim.y,__ipmacc_blockDim.z);
__generated_kernel_region_0<<<__ipmacc_gridDim,__ipmacc_blockDim>>>(
(TYPE *)acc_deviceptr((void*)a),
(TYPE *)acc_deviceptr((void*)c),
(TYPE *)acc_deviceptr((void*)b));
}
/* kernel call statement*/
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Synchronizing the region with host\n");
{
hipError_t err=hipDeviceSynchronize();
if(err!=hipSuccess){
printf("Kernel Launch Error! error code (%d)\n",err);
assert(0&&"Launch Failure!\n");}
}



    }
}
	ipmacc_prompt((char*)"IPMACC: memory copyout c\n");
acc_copyout_and_keep((void*)c,(SIZE+0)*sizeof(TYPE ));



    toc = clock();
    printf(" %6.4f ms\n", (toc - tic) / (TYPE)1000);
  }

  
  
  
  

  

  printf("Calculation on CPU ... ");

  tic = clock();
  for (i = 0; i < LEN; ++i) {
    for (j = 0; j < LEN; j++) {
      TYPE s = 0;
      for (l = 0; l < LEN; l++) {
        s += a [i * LEN + l] * b [l * LEN + j];
      }
      seq [i * LEN + j] = s;
      if (seq [i * LEN + j] != c [i * LEN + j]) {
        fprintf(stderr, "mismatch on %dx%d\n", i, j);
        exit(-1);
      }
    }
  }
  toc = clock();
  printf(" %6.4f ms\n", (toc - tic) / (TYPE)1000);

  fprintf(stderr, "OpenACC matrix multiply test with dynamic arrays was successful!\n");

  return 0;
}


/*__forceinline__*/ __device__ TYPE  __smc_select_0_a(int index1, int index2, TYPE * g_array, TYPE  s_array[16+0+0][16+0+16], int startptr1, int startptr2, int endptr1, int endptr2, int pitch, int diff1, int diff2){
// the pragmas are well-set. do not check the boundaries.
return s_array[index1-startptr1][index2-startptr2];
}
/*__forceinline__*/ __device__ TYPE  __smc_select_0_b(int index1, int index2, TYPE * g_array, TYPE  s_array[16+0+16][16+0+0], int startptr1, int startptr2, int endptr1, int endptr2, int pitch, int diff1, int diff2){
// the pragmas are well-set. do not check the boundaries.
return s_array[index1-startptr1][index2-startptr2];
}

__device__ void __smc_write_0_a(int index1, int index2, TYPE * g_array, TYPE  s_array[16+0+0][16+0+16], int startptr1, int startptr2, int endptr1, int endptr2, int pitch, TYPE  value){
// the pragmas are well-set. do not check the boundaries.
s_array[index1-startptr1][index2-startptr2]=value;
}
__device__ void __smc_write_0_b(int index1, int index2, TYPE * g_array, TYPE  s_array[16+0+16][16+0+0], int startptr1, int startptr2, int endptr1, int endptr2, int pitch, TYPE  value){
// the pragmas are well-set. do not check the boundaries.
s_array[index1-startptr1][index2-startptr2]=value;
}
 __global__ void __generated_kernel_region_0(TYPE * a,TYPE * c,TYPE * b){
int __kernel_getuid_x=threadIdx.x+blockIdx.x*blockDim.x;
int __kernel_getuid_y=threadIdx.y+blockIdx.y*blockDim.y;
int __kernel_getuid_z=threadIdx.z+blockIdx.z*blockDim.z;
int  i;
int  j;
int  l;

/* declare the shared memory of a */
__shared__ TYPE  __kernel_smc_var_data_a[16+0+0][16+0+16];
/*__shared__*/ int __kernel_smc_startpointer_a;
/*__shared__*/ int __kernel_smc_endpointer_a;
/*__shared__*/ int __kernel_smc_startpointer_a_2d;
/*__shared__*/ int __kernel_smc_endpointer_a_2d;
__kernel_smc_endpointer_a=-1;
__kernel_smc_startpointer_a=-1;
__kernel_smc_endpointer_a_2d=-1;
__kernel_smc_startpointer_a_2d=-1;
/*{
int iterator_of_smc=0;
for(iterator_of_smc=threadIdx.x; iterator_of_smc<(16+0+0); iterator_of_smc+=blockDim.x){
//__kernel_smc_var_data_a[iterator_of_smc]=0;
__kernel_smc_var_tag_a[iterator_of_smc]=0;
}
__syncthreads();
}*/

/* declare the shared memory of b */
__shared__ TYPE  __kernel_smc_var_data_b[16+0+16][16+0+0];
/*__shared__*/ int __kernel_smc_startpointer_b;
/*__shared__*/ int __kernel_smc_endpointer_b;
/*__shared__*/ int __kernel_smc_startpointer_b_2d;
/*__shared__*/ int __kernel_smc_endpointer_b_2d;
__kernel_smc_endpointer_b=-1;
__kernel_smc_startpointer_b=-1;
__kernel_smc_endpointer_b_2d=-1;
__kernel_smc_startpointer_b_2d=-1;
/*{
int iterator_of_smc=0;
for(iterator_of_smc=threadIdx.x; iterator_of_smc<(16+0+16); iterator_of_smc+=blockDim.x){
//__kernel_smc_var_data_b[iterator_of_smc]=0;
__kernel_smc_var_tag_b[iterator_of_smc]=0;
}
__syncthreads();
}*/
{
{


      {
{


        {
 i=0+(__kernel_getuid_y);
if( i < LEN)
{
{


            {
 j=0+(__kernel_getuid_x);
if( j < LEN)
{
                TYPE sum = 0;
for(l = 0; l < LEN; l += 16)
{
                  int offseti = l;
                  int offsetj = l;
//go on with the clause (a[0:LEN:0:LEN:FETCH_CHANNEL:i:0:0:offsetj:0:16:false:0:0:0:0],b[0:LEN:0:LEN:FETCH_CHANNEL:offseti:0:16:j:0:0:false:0:0:0:0])
{ // fetch begins

 // FINDING TILE START
__kernel_smc_startpointer_a=i-0-threadIdx.y;
__kernel_smc_startpointer_a_2d=offsetj-0;

 // FINDING DONE

 // FINDING TILE END
bool lastcol=blockIdx.x==(gridDim.x-1);
bool lastrow=blockIdx.y==(gridDim.y-1);
__kernel_smc_endpointer_a=(lastrow)?LEN-1:blockDim.y+__kernel_smc_startpointer_a+0-1;
__kernel_smc_endpointer_a_2d=(lastcol)?LEN-1:blockDim.x+__kernel_smc_startpointer_a_2d+16-1;
// FINDING DONE
//__fusion_merge_boundary_0()
int __ipmacc_length=__kernel_smc_endpointer_a-__kernel_smc_startpointer_a+1;
int __ipmacc_length_2d=__kernel_smc_endpointer_a_2d-__kernel_smc_startpointer_a_2d+1;
int kk=0,kk2=0;
  kk2=threadIdx.x;
  {
   int idx2=__kernel_smc_startpointer_a_2d+kk2;
   if(idx2<(LEN) && idx2>=(0))
   {
for(kk=threadIdx.y; kk<__ipmacc_length; kk+=blockDim.x)
{
 int idx=__kernel_smc_startpointer_a+kk;
 if(idx<(LEN) && idx>=(0))
 {
__kernel_smc_var_data_a[kk][kk2]=a[idx*LEN+idx2];
//__kernel_smc_var_tag_a[kk][kk2]=1;
//__fusion_merge_fetch_0()
   }
  }
 }
}
__syncthreads();
} // end of fetch
#define a(index) __smc_select_0_a(index, a, __kernel_smc_var_data_a, __kernel_smc_startpointer_a, __kernel_smc_startpointer_a_2d, LEN)

// 1 unique indexes
// [0] i*LEN+m
	#define __ipmacc_smc_index_a_0_dim1 i-__kernel_smc_startpointer_a
	#define __ipmacc_smc_index_a_0_dim2 m-__kernel_smc_startpointer_a_2d
{ // fetch begins

 // FINDING TILE START
__kernel_smc_startpointer_b=offseti-0;
__kernel_smc_startpointer_b_2d=j-0-threadIdx.x;

 // FINDING DONE

 // FINDING TILE END
bool lastcol=blockIdx.x==(gridDim.x-1);
bool lastrow=blockIdx.y==(gridDim.y-1);
__kernel_smc_endpointer_b=(lastrow)?LEN-1:blockDim.y+__kernel_smc_startpointer_b+16-1;
__kernel_smc_endpointer_b_2d=(lastcol)?LEN-1:blockDim.x+__kernel_smc_startpointer_b_2d+0-1;
// FINDING DONE
//__fusion_merge_boundary_2()
int __ipmacc_length=__kernel_smc_endpointer_b-__kernel_smc_startpointer_b+1;
int __ipmacc_length_2d=__kernel_smc_endpointer_b_2d-__kernel_smc_startpointer_b_2d+1;
int kk=0,kk2=0;
  for(kk2=threadIdx.x; kk2<__ipmacc_length_2d; kk2+=blockDim.y)
  {
   int idx2=__kernel_smc_startpointer_b_2d+kk2;
   if(idx2<(LEN) && idx2>=(0))
   {
  kk=threadIdx.y;
{
 int idx=__kernel_smc_startpointer_b+kk;
 if(idx<(LEN) && idx>=(0))
 {
__kernel_smc_var_data_b[kk][kk2]=b[idx*LEN+idx2];
//__kernel_smc_var_tag_b[kk][kk2]=1;
//__fusion_merge_fetch_2()
   }
  }
 }
}
__syncthreads();
} // end of fetch
#define b(index) __smc_select_0_b(index, b, __kernel_smc_var_data_b, __kernel_smc_startpointer_b, __kernel_smc_startpointer_b_2d, LEN)

// 1 unique indexes
// [0] m*LEN+j
	#define __ipmacc_smc_index_b_0_dim1 m-__kernel_smc_startpointer_b
	#define __ipmacc_smc_index_b_0_dim2 j-__kernel_smc_startpointer_b_2d

{


                  {
                    if (j < LEN && i < LEN) {
                      int m;
for(m = l; m < MIN(l + 16, LEN); m++)
{
                        sum += __kernel_smc_var_data_a[__ipmacc_smc_index_a_0_dim1][__ipmacc_smc_index_a_0_dim2] /* replacing a [i * LEN + m]*/  * __kernel_smc_var_data_b[__ipmacc_smc_index_b_0_dim1][__ipmacc_smc_index_b_0_dim2] /* replacing b [m * LEN + j]*/ ;
                      }
}
                  }
}
#undef a
#undef b

//end up with the clause (a[0:LEN:0:LEN:FETCH_CHANNEL:i:0:0:offsetj:0:16:false:0:0:0:0],b[0:LEN:0:LEN:FETCH_CHANNEL:offseti:0:16:j:0:0:false:0:0:0:0])
}
if (j < LEN && i < LEN) {
                  c [i * LEN + j] = sum;
                }
              }

}
}
}

}
}
}
}
}
//append writeback of scalar variables
}

