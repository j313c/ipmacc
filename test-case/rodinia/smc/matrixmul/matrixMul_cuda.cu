#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <openacc.h>
#define IPMACC_MAX1(A)   (A)
#define IPMACC_MAX2(A,B) (A>B?A:B)
#define IPMACC_MAX3(A,B,C) (A>B?(A>C?A:(B>C?B:C)):(B>C?C:B))
#ifdef __cplusplus
#include "openacc_container.h"
#endif

#include <hip/hip_runtime.h>

#include <malloc.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>

#include <math.h>

//#define LEN 1024
#define SIZE LEN * LEN

#define TYPE double
#define MIN(a, b)    (a < b ? a : b)


__global__ void __generated_kernel_region_0(TYPE * a,TYPE * c,TYPE * b, int LEN);

int main(int argc, char *argv[])
{
    int i;
#ifdef __NVCUDA__
    acc_init(acc_device_nvcuda);
#endif
#ifdef __NVOPENCL__
    acc_init(acc_device_nvocl);
#endif

    int LEN = -1;
    if(argc!=2){
        printf("usage: ./matMul <size>\n");
        exit(-1);
    }else{
        sscanf(argv[1], "%d", &LEN);
    }
    assert(LEN>0);



    TYPE *a, *b, *c;

    TYPE *seq;

    a = (TYPE*)malloc(SIZE * sizeof(TYPE));
    b = (TYPE*)malloc(SIZE * sizeof(TYPE));
    c = (TYPE*)malloc(SIZE * sizeof(TYPE));
    seq = (TYPE*)malloc(SIZE * sizeof(TYPE));


    for (i = 0; i < SIZE; ++i) {

        a [i] = (TYPE)i;
        b [i] = (TYPE)2 * i;
        c [i] = 0.0f;
    }  

    unsigned long long int tic, toc;

    int k, j, l;
    for (k = 0; k < 3; k++) {
        printf("Calculation on GPU ... ");
        tic = clock();



        ipmacc_prompt((char*)"IPMACC: memory allocation c\n");
        acc_present_or_create((void*)c,(SIZE+0)*sizeof(TYPE ));
        ipmacc_prompt((char*)"IPMACC: memory allocation a\n");
        acc_present_or_create((void*)a,(SIZE+0)*sizeof(TYPE ));
        ipmacc_prompt((char*)"IPMACC: memory allocation b\n");
        acc_present_or_create((void*)b,(SIZE+0)*sizeof(TYPE ));
        ipmacc_prompt((char*)"IPMACC: memory copyin c\n");
        acc_pcopyin((void*)c,(SIZE+0)*sizeof(TYPE ));
        ipmacc_prompt((char*)"IPMACC: memory copyin a\n");
        acc_pcopyin((void*)a,(SIZE+0)*sizeof(TYPE ));
        ipmacc_prompt((char*)"IPMACC: memory copyin b\n");
        acc_pcopyin((void*)b,(SIZE+0)*sizeof(TYPE ));


        {


            {



                /* kernel call statement [0, -1]*/
                {
                    dim3 __ipmacc_gridDim(1,1,1);
                    dim3 __ipmacc_blockDim(1,1,1);
                    __ipmacc_blockDim.x=16;
                    __ipmacc_gridDim.x=(((abs((int)((LEN))-(0+0)))/(1))/__ipmacc_blockDim.x)+(((((abs((int)((LEN))-(0+0)))/(1))%(16))==0?0:1));
                    __ipmacc_blockDim.y=16;
                    __ipmacc_gridDim.y=(((abs((int)((LEN))-(0+0)))/(1))/__ipmacc_blockDim.y)+(((((abs((int)((LEN))-(0+0)))/(1))%(16))==0?0:1));
                    if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Launching kernel 0 > gridDim: (%u,%u,%u)\tblockDim: (%u,%u,%u)\n",__ipmacc_gridDim.x,__ipmacc_gridDim.y,__ipmacc_gridDim.z,__ipmacc_blockDim.x,__ipmacc_blockDim.y,__ipmacc_blockDim.z);
                    __generated_kernel_region_0<<<__ipmacc_gridDim,__ipmacc_blockDim>>>(
                            (TYPE *)acc_deviceptr((void*)a),
                            (TYPE *)acc_deviceptr((void*)c),
                            (TYPE *)acc_deviceptr((void*)b),
                            LEN);
                }
                /* kernel call statement*/
                if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Synchronizing the region with host\n");
                {
                    hipError_t err=hipDeviceSynchronize();
                    if(err!=hipSuccess){
                        printf("Kernel Launch Error! error code (%d)\n",err);
                        assert(0&&"Launch Failure!\n");}
                }



            }
        }
        ipmacc_prompt((char*)"IPMACC: memory copyout c\n");
        acc_copyout_and_keep((void*)c,(SIZE+0)*sizeof(TYPE ));



        toc = clock();
        printf(" %6.4f ms\n", (toc - tic) / (TYPE)1000);
    }







    /*
    printf("Calculation on CPU ... ");

    tic = clock();
    for (i = 0; i < LEN; ++i) {
        for (j = 0; j < LEN; j++) {
            TYPE s = 0;
            for (l = 0; l < LEN; l++) {
                s += a [i * LEN + l] * b [l * LEN + j];
            }
            seq [i * LEN + j] = s;
            if (seq [i * LEN + j] != c [i * LEN + j]) {
                fprintf(stderr, "mismatch on %dx%d\n", i, j);
                exit(-1);
            }
        }
    }
    toc = clock();
    printf(" %6.4f ms\n", (toc - tic) / (TYPE)1000);

    fprintf(stderr, "OpenACC matrix multiply test with dynamic arrays was successful!\n");
    */
    return 0;
}


/*__forceinline__*/ __device__ TYPE  __smc_select_0_a(int index1, int index2, TYPE * g_array, TYPE  s_array[16+0+0][16+0+16], int startptr1, int startptr2, int endptr1, int endptr2, int pitch, int diff1, int diff2){
    // the pragmas are well-set. do not check the boundaries.
    return s_array[index1-startptr1][index2-startptr2];
}
/*__forceinline__*/ __device__ TYPE  __smc_select_0_b(int index1, int index2, TYPE * g_array, TYPE  s_array[16+0+16][16+0+0], int startptr1, int startptr2, int endptr1, int endptr2, int pitch, int diff1, int diff2){
    // the pragmas are well-set. do not check the boundaries.
    return s_array[index1-startptr1][index2-startptr2];
}

__device__ void __smc_write_0_a(int index1, int index2, TYPE * g_array, TYPE  s_array[16+0+0][16+0+16], int startptr1, int startptr2, int endptr1, int endptr2, int pitch, TYPE  value){
    // the pragmas are well-set. do not check the boundaries.
    s_array[index1-startptr1][index2-startptr2]=value;
}
__device__ void __smc_write_0_b(int index1, int index2, TYPE * g_array, TYPE  s_array[16+0+16][16+0+0], int startptr1, int startptr2, int endptr1, int endptr2, int pitch, TYPE  value){
    // the pragmas are well-set. do not check the boundaries.
    s_array[index1-startptr1][index2-startptr2]=value;
}
__global__ void __generated_kernel_region_0(TYPE * a,TYPE * c,TYPE * b, int LEN){
    int __kernel_getuid_x=threadIdx.x+blockIdx.x*blockDim.x;
    int __kernel_getuid_y=threadIdx.y+blockIdx.y*blockDim.y;
    int __kernel_getuid_z=threadIdx.z+blockIdx.z*blockDim.z;
    int  i;
    int  j;
    int  l;

    /* declare the shared memory of a */
    __shared__ TYPE  __kernel_smc_var_data_a[16][16];
    __shared__ TYPE  __kernel_smc_var_data_b[16][16];
    i=0+(__kernel_getuid_y);
    if( i < LEN)
    {
        {
            {
                j=0+(__kernel_getuid_x);
                if( j < LEN)
                {
                    TYPE sum = 0;
                    for(l = 0; l < LEN; l += 16)
                    {
                        int offseti = l;
                        int offsetj = l;
                        {
                            __syncthreads();
                            {
                                int idx2 = offsetj + threadIdx.x;
                                int idx  = i - threadIdx.y + threadIdx.y;
                                __kernel_smc_var_data_a[threadIdx.y][threadIdx.x]=a[idx*LEN+idx2];
                            }
                            {
                                int idx  = offseti + threadIdx.y;
                                int idx2 = j - threadIdx.x + threadIdx.x;
                                __kernel_smc_var_data_b[threadIdx.y][threadIdx.x]=b[idx*LEN+idx2];
                            }
                            __syncthreads();

                            {
                                int m;
                                //for(m = l; m < MIN(l + 16, LEN); m++)
                                for(m = 0; m < 16; m++)
                                {
                                    sum += __kernel_smc_var_data_a[threadIdx.y][m] /* replacing a [i * LEN + m]*/  * __kernel_smc_var_data_b[m][threadIdx.x] /* replacing b [m * LEN + j]*/ ;
                                }
                            }
                        }

                        //end up with the clause (a[0:LEN:0:LEN:FETCH_CHANNEL:i:0:0:offsetj:0:16:false:0:0:0:0],b[0:LEN:0:LEN:FETCH_CHANNEL:offseti:0:16:j:0:0:false:0:0:0:0])
                    }
                    if (j < LEN && i < LEN) {
                        c [i * LEN + j] = sum;
                    }
                }
            }
        }
    }
    //append writeback of scalar variables
}

