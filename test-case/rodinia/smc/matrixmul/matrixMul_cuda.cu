#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <openacc.h>

#include <hip/hip_runtime.h>

#include <malloc.h>
#include <time.h>
#include <math.h>

#define SIZE LEN * LEN

#define TYPE double

__global__ void __generated_kernel_region_0(TYPE * a,TYPE * c,TYPE * b, int LEN);

int main(int argc, char *argv[])
{
    int i;
    int LEN = -1;
    if(argc!=2){
        printf("usage: ./matMul <size>\n");
        exit(-1);
    }else{
        sscanf(argv[1], "%d", &LEN);
    }
    assert(LEN>0);



    TYPE *a, *b, *c;;;

    TYPE *seq;

    a = (TYPE*)malloc(SIZE * sizeof(TYPE));
    b = (TYPE*)malloc(SIZE * sizeof(TYPE));
    c = (TYPE*)malloc(SIZE * sizeof(TYPE));
    seq = (TYPE*)malloc(SIZE * sizeof(TYPE));


    for (i = 0; i < SIZE; ++i) {

        a [i] = (TYPE)i;
        b [i] = (TYPE)2 * i;
        c [i] = 0.0f;
    }  

    unsigned long long int tic, toc;;

    int k, j, l;
    for (k = 0; k < 3; k++) {
        printf("Calculation on GPU ... ");
        tic = clock();

        acc_present_or_create((void*)c,(SIZE+0)*sizeof(TYPE ));
        acc_present_or_create((void*)a,(SIZE+0)*sizeof(TYPE ));
        acc_present_or_create((void*)b,(SIZE+0)*sizeof(TYPE ));
        acc_pcopyin((void*)c,(SIZE+0)*sizeof(TYPE ));
        acc_pcopyin((void*)a,(SIZE+0)*sizeof(TYPE ));
        acc_pcopyin((void*)b,(SIZE+0)*sizeof(TYPE ));


        {
            {
                /* kernel call statement [0, -1]*/
                {
                    dim3 __ipmacc_gridDim(1,1,1);
                    dim3 __ipmacc_blockDim(1,1,1);
                    __ipmacc_blockDim.x=16;
                    __ipmacc_gridDim.x=(((abs((int)((LEN))-(0+0)))/(1))/__ipmacc_blockDim.x)+(((((abs((int)((LEN))-(0+0)))/(1))%(16))==0?0:1));
                    __ipmacc_blockDim.y=16;
                    __ipmacc_gridDim.y=(((abs((int)((LEN))-(0+0)))/(1))/__ipmacc_blockDim.y)+(((((abs((int)((LEN))-(0+0)))/(1))%(16))==0?0:1));
                    if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Launching kernel 0 > gridDim: (%u,%u,%u)\tblockDim: (%u,%u,%u)\n",__ipmacc_gridDim.x,__ipmacc_gridDim.y,__ipmacc_gridDim.z,__ipmacc_blockDim.x,__ipmacc_blockDim.y,__ipmacc_blockDim.z);
                    __generated_kernel_region_0<<<__ipmacc_gridDim,__ipmacc_blockDim>>>(
                            (TYPE *)acc_deviceptr((void*)a),
                            (TYPE *)acc_deviceptr((void*)c),
                            (TYPE *)acc_deviceptr((void*)b),
                            LEN);
                }
                /* kernel call statement*/
                {
                    hipError_t err=hipDeviceSynchronize();
                    if(err!=hipSuccess){
                        printf("Kernel Launch Error! error code (%d)\n",err);
                        exit(-1);
                    }
                }



            }
        }
        ipmacc_prompt((char*)"IPMACC: memory copyout c\n");
        acc_copyout_and_keep((void*)c,(SIZE+0)*sizeof(TYPE ));



        toc = clock();
        printf(" %6.4f ms\n", (toc - tic) / (TYPE)1000);
    }







    /*
    printf("Calculation on CPU ... ");

    tic = clock();
    for (i = 0; i < LEN; ++i) {
        for (j = 0; j < LEN; j++) {
            TYPE s = 0;
            for (l = 0; l < LEN; l++) {
                s += a [i * LEN + l] * b [l * LEN + j];
            }
            seq [i * LEN + j] = s;
            if (seq [i * LEN + j] != c [i * LEN + j]) {
                fprintf(stderr, "mismatch on %dx%d\n", i, j);
                exit(-1);
            }
        }
    }
    toc = clock();
    printf(" %6.4f ms\n", (toc - tic) / (TYPE)1000);

    fprintf(stderr, "OpenACC matrix multiply test with dynamic arrays was successful!\n");
    */
    return 0;
}


__global__ void __generated_kernel_region_0(TYPE * a,TYPE * c,TYPE * b, int LEN){
    int __kernel_getuid_x=threadIdx.x+blockIdx.x*blockDim.x;
    int __kernel_getuid_y=threadIdx.y+blockIdx.y*blockDim.y;
    int  i;
    int  j;
    int  l;

    /* declare the shared memory of a */
    __shared__ TYPE  __kernel_smc_var_data_a[16][16];
    __shared__ TYPE  __kernel_smc_var_data_b[16][16];
    i=0+(__kernel_getuid_y);
    if( i < LEN)
    {
        {
            {
                j=0+(__kernel_getuid_x);
                if( j < LEN)
                {
                    TYPE sum = 0;
                    for(l = 0; l < LEN; l += 16)
                    {
                        int offseti = l;
                        int offsetj = l;
                        {
                            __syncthreads();
                            {
                                int idx2 = offsetj + threadIdx.x;
                                int idx  = i - threadIdx.y + threadIdx.y;
                                __kernel_smc_var_data_a[threadIdx.y][threadIdx.x]=a[idx*LEN+idx2];
                            }
                            {
                                int idx  = offseti + threadIdx.y;
                                int idx2 = j - threadIdx.x + threadIdx.x;
                                __kernel_smc_var_data_b[threadIdx.y][threadIdx.x]=b[idx*LEN+idx2];
                            }
                            __syncthreads();

                            {
                                int m;
                                for(m = 0; m < 16; m++)
                                {
                                    sum += __kernel_smc_var_data_a[threadIdx.y][m] /* replacing a [i * LEN + m]*/  * __kernel_smc_var_data_b[m][threadIdx.x] /* replacing b [m * LEN + j]*/ ;
                                }
                            }
                        }

                        //end up with the clause (a[0:LEN:0:LEN:FETCH_CHANNEL:i:0:0:offsetj:0:16:false:0:0:0:0],b[0:LEN:0:LEN:FETCH_CHANNEL:offseti:0:16:j:0:0:false:0:0:0:0])
                    }
                    if (j < LEN && i < LEN) {
                        c [i * LEN + j] = sum;
                    }
                }
            }
        }
    }
    //append writeback of scalar variables
}

