#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <openacc.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <openacc.h>
#include <math.h>

#if defined(_WIN32) || defined(_WIN64)
#include <sys/timeb.h>
#define gettime(a) _ftime(a)
#define usec(t1, t2) ((((t2).time - (t1).time) * 1000 + ((t2).millitm - (t1).millitm)) * 100)
typedef struct _timeb timestruct;
#else
#include <sys/time.h>
#define gettime(a) gettimeofday(a, NULL)
#define usec(t1, t2) (((t2).tv_sec - (t1).tv_sec) * 1000000 + ((t2).tv_usec - (t1).tv_usec))
typedef struct timeval timestruct;
#endif

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))

__global__ void __generated_kernel_region_0(float * a,int  m,int  n,float * b,float  w2,float  w1,float  w0);

void smooth(float* a, float* b, float w0, float w1, float w2, int n, int m, int niters)
{
    int i, j, iter;;;
    float* tmp;
    for (iter = 1; iter <= niters; ++iter) {


        acc_create((void*)a,(n*m+0)*sizeof(float ));
        acc_create((void*)b,(n*m+0)*sizeof(float ));
        acc_copyin((void*)a,(n*m+0)*sizeof(float ));
        acc_copyin((void*)b,(n*m+0)*sizeof(float ));

        /* kernel call statement [0]*/
        {
            dim3 __ipmacc_gridDim(1,1,1);
            dim3 __ipmacc_blockDim(1,1,1);
            __ipmacc_blockDim.x=16;
            __ipmacc_gridDim.x=(((abs((int)(((m-1)))-(1+0)))/(1))/__ipmacc_blockDim.x)+(((((abs((int)(((m-1)))-(1+0)))/(1))%(16))==0?0:1));
            __ipmacc_blockDim.y=16;
            __ipmacc_gridDim.y=(((abs((int)(((n-1)))-(1+0)))/(1))/__ipmacc_blockDim.y)+(((((abs((int)(((n-1)))-(1+0)))/(1))%(16))==0?0:1));
            if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Launching kernel 0 > gridDim: (%u,%u,%u)\tblockDim: (%u,%u,%u)\n",__ipmacc_gridDim.x,__ipmacc_gridDim.y,__ipmacc_gridDim.z,__ipmacc_blockDim.x,__ipmacc_blockDim.y,__ipmacc_blockDim.z);


            int borderCols = 1;
            int borderRows = 1;
            int smallBlockCol = 16-2;
            int smallBlockRow = 16-2;
            int blockCols = m/smallBlockCol+((m%smallBlockCol==0)?0:1);
            int blockRows = n/smallBlockRow+((n%smallBlockRow==0)?0:1);
            dim3 gridDimen(blockRows, blockCols);


            __generated_kernel_region_0<<<gridDimen,__ipmacc_blockDim>>>(
                    (float *)acc_deviceptr((void*)a),
                    m,
                    n,
                    (float *)acc_deviceptr((void*)b),
                    w2,
                    w1,
                    w0);
        }
        /* kernel call statement*/
        acc_copyout_and_keep((void*)a,(n*m+0)*sizeof(float ));
        {
            hipError_t err=hipDeviceSynchronize();
            if(err!=hipSuccess){
                printf("Kernel Launch Error! error code (%d)\n",err);
                exit(-1);
            }
        }

        tmp = a;  a = b;  b = tmp;
    }
}

void smoothhost(float* a, float* b, float w0, float w1, float w2, int n, int m, int niters)
{
    int i, j, iter;;;
    float* tmp;
    for (iter = 1; iter <= niters; ++iter) {
        for (i = 1; i < n - 1; ++i) {
            for (j = 1; j < m - 1; ++j) {
                a [i * m + j] = w0 * b [i * m + j] +
                    w1 * (b [(i - 1) * m + j] + b [(i + 1) * m + j] + b [i * m + j - 1] + b [i * m + j + 1]) +
                    w2 * (b [(i - 1) * m + j - 1] + b [(i - 1) * m + j + 1] + b [(i + 1) * m + j - 1] + b [(i + 1) * m + j + 1]);
            }
        }
        tmp = a;  a = b;  b = tmp;
    }
}

void doprt(char* s, float* a, float* ah, int i, int j, int n, int m)
{
    printf("%s[%d][%d] = %g  =  %g\n", s, i, j, a [i * m + j], ah [i * m + j]);
}

int main(int argc, char* argv[])
{
    float *aa, *bb, *aahost, *bbhost;;;;
    int i, j;;
    float w0, w1, w2;;;
    int n, m, aerrs, berrs, iters;;;;
    float dif, rdif, tol;;;
    timestruct t1, t2, t3;;;
    long long cgpu, chost;;

    n = 1024;
    m = 1024;
    iters = 1;

    if( argc > 1 ){
        n = atoi( argv[1] );
        if( argc > 2 ){
            m = atoi( argv[2] );
            if( argc > 3 ){
                iters = atoi( argv[3] );
            }
        }
    }


    if (n <= 0) {
        n = 1000;
    }
    if (m <= 0) {
        m = n;
    }
    if (iters <= 0) {
        iters = 10;
    }

    aa = (float*)malloc(sizeof(float) * n * m);
    aahost = (float*)malloc(sizeof(float) * n * m);
    bb = (float*)malloc(sizeof(float) * n * m);
    bbhost = (float*)malloc(sizeof(float) * n * m);
    for (i = 0; i < n; ++i) {
        for (j = 0; j < m; ++j) {
            aa [i * m + j] = 0;
            aahost [i * m + j] = 0;
            bb [i * m + j] = i * 1000 + j;
            bbhost [i * m + j] = i * 1000 + j;
        }
    }
    w0 = 0.5;
    w1 = 0.3;
    w2 = 0.2;
    gettime(&t1);
    smooth(aa, bb, w0, w1, w2, n, m, iters);
    gettime(&t2);
    smoothhost(aahost, bbhost, w0, w1, w2, n, m, iters);
    gettime(&t3);

    cgpu = usec(t1, t2);
    chost = usec(t2, t3);

    printf("matrix %d x %d, %d iterations\n", n, m, iters);
    printf("%13ld microseconds optimized\n", cgpu);
    printf("%13ld microseconds on host\n", chost);

    aerrs = berrs = 0;
    tol = 0.000005;
    for (i = 0; i < n; ++i) {
        for (j = 0; j < m; ++j) {
            rdif = dif = fabsf(aa [i * m + j] - aahost [i * m + j]);
            if (aahost [i * m + j]) {
                rdif = fabsf(dif / aahost [i * m + j]);
            }
            if (rdif > tol) {
                ++aerrs;
                if (aerrs < 10) {
                    printf("aa[%d][%d] = %12.7e != %12.7e, dif=%12.7e\n", i, j, (double)aa [i * m + j], (double)aahost [i * m + j], (double)dif);
                }
            }
            rdif = dif = fabsf(bb [i * m + j] - bbhost [i * m + j]);
            if (bbhost [i * m + j]) {
                rdif = fabsf(dif / bbhost [i * m + j]);
            }
            if (rdif > tol) {
                ++berrs;
                if (berrs < 10) {
                    printf("bb[%d][%d] = %12.7e != %12.7e, dif=%12.7e\n", i, j, (double)bb [i * m + j], (double)bbhost [i * m + j], (double)dif);
                }
            }
        }
    }
    if (aerrs == 0 && berrs == 0) {
        fprintf(stderr, "no errors found\n");
        return 0;
    }else{
        fprintf(stderr, "%d ERRORS found\n", aerrs + berrs);
        return 1;
    }
}



__global__ void __generated_kernel_region_0(float * a,int  m,int  n,float * b,float  w2,float  w1,float  w0){
    int __kernel_getuid_x=threadIdx.x+blockIdx.x*blockDim.x;
    int __kernel_getuid_y=threadIdx.y+blockIdx.y*blockDim.y;
    int  i;
    int  j;

    __shared__ float btile[16+2][16+2];

    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx=threadIdx.x;
    int ty=threadIdx.y;


    // each block finally computes result for a small block
    // after N iterations. 
    // it is the non-overlapping small blocks that cover 
    // all the input data

    // calculate the small block size
    int iteration=1;
    int small_block_rows = 16-iteration*2;//EXPAND_RATE
    int small_block_cols = 16-iteration*2;//EXPAND_RATE

    // calculate the boundary for the block according to 
    // the boundary of its small block
    int blkY = small_block_rows*by-iteration;
    int blkX = small_block_cols*bx-iteration;
    int blkYmax = blkY+16-1;
    int blkXmax = blkX+16-1;

    // calculate the global thread coordination
    int yidx = blkY+ty;
    int xidx = blkX+tx;

    // load data if it is within the valid input range
    int loadYidx=yidx, loadXidx=xidx;
    int index = m*loadYidx+loadXidx;

    if(IN_RANGE(loadYidx, 0, n-1) && IN_RANGE(loadXidx, 0, m-1)){
        btile[ty][tx] = b[index];  // Load the temperature data from global memory to shared memory
    }
    __syncthreads();

    if((threadIdx.x>0 && threadIdx.x<(blockDim.x-1)) && 
       (threadIdx.y>0 && threadIdx.y<(blockDim.y-1)) &&
       (xidx>0 && xidx<(m-1)) &&
       (yidx>0 && yidx<(n-1))){
        //a [i * m + j] = w0 * b [i * m + j] + \
            w1 * (b [(i - 1) * m + j] + b [(i + 1) * m + j] + b [i * m + j - 1] + b [i * m + j + 1]) + \
            w2 * (b [(i - 1) * m + j - 1] + b [(i - 1) * m + j + 1] + b [(i + 1) * m + j - 1] + b [(i + 1) * m + j + 1])
        a [yidx * m + xidx] = w0 * btile[ty][tx] +
            w1 * (btile[ty-1][tx] + btile[ty+1][tx] + btile[ty][tx-1] + btile[ty][tx+1]) +
            w2 * (btile[ty-1][tx-1] + btile[ty-1][tx+1] + btile[ty+1][tx-1] + btile[ty+1][tx+1]);
    }
}

