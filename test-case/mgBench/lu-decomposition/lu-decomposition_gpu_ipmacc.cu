#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <openacc.h>
#define IPMACC_MAX1(A)   (A)
#define IPMACC_MAX2(A,B) (A>B?A:B)
#define IPMACC_MAX3(A,B,C) (A>B?(A>C?A:(B>C?B:C)):(B>C?C:B))
#include <hip/hip_runtime.h>



#include <stdio.h>
#include <openacc.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <limits.h>

#define SIZE 510

float a[SIZE * SIZE];
float b[SIZE * SIZE];

FILE *fil;
FILE *out;


void
init (int s)
{
  int i, j, q;
  q = SIZE * SIZE;
  for (i = 0; i < s; ++i)
    {
      for (j = 0; j < s; ++j)
	{
	  a[i * s + j] = (float) ((s * s) - (i + j));
	  b[i * s + j] = 0.0f;
	}
    }
}



__global__ void __generated_kernel_region_0 (float *a, float *b, int s,
					     float sum);

void
Crout_GPU (int s)
{
  int k, j, i;
  float sum;
  double start, finish, elapsed;
  start = (double) clock () / CLOCKS_PER_SEC;



  ipmacc_prompt ((char *) "IPMACC: memory allocation b\n");
  acc_present_or_create ((void *) b, SIZE * SIZE * sizeof (float));
  ipmacc_prompt ((char *) "IPMACC: memory allocation a\n");
  acc_present_or_create ((void *) a, SIZE * SIZE * sizeof (float));
  ipmacc_prompt ((char *) "IPMACC: memory copyin b\n");
  acc_pcopyin ((void *) b, SIZE * SIZE * sizeof (float));
  ipmacc_prompt ((char *) "IPMACC: memory copyin a\n");
  acc_pcopyin ((void *) a, SIZE * SIZE * sizeof (float));


  {


    {


      ipmacc_prompt ((char *) "IPMACC: memory allocation b\n");
      acc_present_or_create ((void *) b, SIZE * SIZE * sizeof (float));
      ipmacc_prompt ((char *) "IPMACC: memory allocation a\n");
      acc_present_or_create ((void *) a, SIZE * SIZE * sizeof (float));
      ipmacc_prompt ((char *) "IPMACC: memory copyin b\n");
      acc_pcopyin ((void *) b, SIZE * SIZE * sizeof (float));
      ipmacc_prompt ((char *) "IPMACC: memory copyin a\n");
      acc_pcopyin ((void *) a, SIZE * SIZE * sizeof (float));

/* kernel call statement [0, 1]*/
      if (getenv ("IPMACC_VERBOSE"))
	printf ("IPMACC: Launching kernel 0 > gridDim: %d\tblockDim: %d\n",
		(((abs (((s)) - 0)) / (1))) / 256 + 1, 256);
      __generated_kernel_region_0 <<< (((abs (((s)) - 0)) / (1))) / 256 + 1,
	256 >>> ((float *) acc_deviceptr ((void *) a),
		 (float *) acc_deviceptr ((void *) b), s, sum);
/* kernel call statement*/
      ipmacc_prompt ((char *) "IPMACC: memory copyout b\n");
      acc_copyout_and_keep ((void *) b, SIZE * SIZE * sizeof (float));
      if (getenv ("IPMACC_VERBOSE"))
	printf ("IPMACC: Synchronizing the region with host\n");
      hipDeviceSynchronize ();



    }
  }
  ipmacc_prompt ((char *) "IPMACC: memory copyout b\n");
  acc_copyout_and_keep ((void *) b, SIZE * SIZE * sizeof (float));




  finish = (double) clock () / CLOCKS_PER_SEC;
  elapsed = finish - start;
  fprintf (fil, "%.6lf;", elapsed);
}



void
print_matrix (int s)
{
  int i, j;
  for (i = 0; i < s; i++)
    {
      for (j = 0; j < s; j++)
	{
	  if (b[i * s + j] == INT_MAX)
	    {
	      continue;
	    }
	  fprintf (out, "%.6f ", b[i * s + j]);
	}
      fprintf (out, "\n");
    }
}

int
main (int argc, char *argv[])
{
  int i;
  int points, var;
  points = atoi (argv[1]);
  var = SIZE / points;

  fil = fopen ("time_gpu.csv", "w+");
  out = fopen ("result_gpu.txt", "w+");
  fprintf (fil, "SIZE,LU decomposition GPU,\n");
  for (i = 2; i < SIZE; i += var)
    {
      init (i);
      fprintf (fil, "%d,", i);
      Crout_GPU (i);
      fprintf (fil, "\n");
    }
  fclose (fil);
  fclose (out);
  return 0;
}



__global__ void
__generated_kernel_region_0 (float *a, float *b, int s, float sum)
{
  int __kernel_getuid = threadIdx.x + blockIdx.x * blockDim.x;
  int i;
  int k;
  int j;
  {
    {


      {
	{


	  {
	    k = 0 + (__kernel_getuid);
	    if (k < s)
	      {
		for (j = k; j < s; ++j)
		  {
		    sum = 0.0;
		    for (i = 0; i < k; ++i)
		      {
			sum += b[j * s + i] * b[i * s + k];
		      }
		    b[j * s + k] = (a[j * s + k] - sum);
		  }

		for (i = k + 1; i < s; ++i)
		  {
		    sum = 0.0;
		    for (j = 0; j < k; ++j)
		      {
			sum += b[k * s + j] * b[i * s + i];
		      }
		    b[k * s + i] = (a[k * s + i] - sum) / b[k * s + k];
		  }
	      }

	  }
	}
      }
    }
  }
}
