#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <openacc.h>
#define IPMACC_MAX1(A)   (A)
#define IPMACC_MAX2(A,B) (A>B?A:B)
#define IPMACC_MAX3(A,B,C) (A>B?(A>C?A:(B>C?B:C)):(B>C?C:B))
#include <hip/hip_runtime.h>



#include <stdio.h>
#include <openacc.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#define SIZE 1210

float a[SIZE * SIZE];
float b[SIZE * SIZE];
float c[SIZE * SIZE];

FILE *fil;
FILE *out;


void
init (int s)
{
  int i, j;
  for (i = 0; i < s; ++i)
    {
      for (j = 0; j < s; ++j)
	{
	  a[i * s + j] = (float) i + j;
	  b[i * s + j] = (float) i - j;
	  c[i * s + j] = 0.0f;
	}
    }
}



__global__ void __generated_kernel_region_0 (float *a, float *c, float *b,
					     int s);

void
sum_GPU (int s)
{
  int i, j;
  float start, finish, elapsed;
  start = (float) clock () / (CLOCKS_PER_SEC * 1000);


  ipmacc_prompt ((char *) "IPMACC: memory allocation c\n");
  acc_create ((void *) c, SIZE * SIZE * sizeof (float));
  ipmacc_prompt ((char *) "IPMACC: memory allocation a\n");
  acc_create ((void *) a, SIZE * SIZE * sizeof (float));
  ipmacc_prompt ((char *) "IPMACC: memory allocation b\n");
  acc_create ((void *) b, SIZE * SIZE * sizeof (float));
  ipmacc_prompt ((char *) "IPMACC: memory copyin c\n");
  acc_copyin ((void *) c, SIZE * SIZE * sizeof (float));
  ipmacc_prompt ((char *) "IPMACC: memory copyin a\n");
  acc_copyin ((void *) a, SIZE * SIZE * sizeof (float));
  ipmacc_prompt ((char *) "IPMACC: memory copyin b\n");
  acc_copyin ((void *) b, SIZE * SIZE * sizeof (float));


  {


    {


      ipmacc_prompt ((char *) "IPMACC: memory allocation c\n");
      acc_create ((void *) c, SIZE * SIZE * sizeof (float));
      ipmacc_prompt ((char *) "IPMACC: memory allocation a\n");
      acc_create ((void *) a, SIZE * SIZE * sizeof (float));
      ipmacc_prompt ((char *) "IPMACC: memory allocation b\n");
      acc_create ((void *) b, SIZE * SIZE * sizeof (float));
      ipmacc_prompt ((char *) "IPMACC: memory copyin c\n");
      acc_copyin ((void *) c, SIZE * SIZE * sizeof (float));
      ipmacc_prompt ((char *) "IPMACC: memory copyin a\n");
      acc_copyin ((void *) a, SIZE * SIZE * sizeof (float));
      ipmacc_prompt ((char *) "IPMACC: memory copyin b\n");
      acc_copyin ((void *) b, SIZE * SIZE * sizeof (float));

/* kernel call statement [0, 1]*/
      if (getenv ("IPMACC_VERBOSE"))
	printf ("IPMACC: Launching kernel 0 > gridDim: %d\tblockDim: %d\n",
		(((abs (((s)) - 0)) / (1))) / 256 + 1, 256);
      __generated_kernel_region_0 <<< (((abs (((s)) - 0)) / (1))) / 256 + 1,
	256 >>> ((float *) acc_deviceptr ((void *) a),
		 (float *) acc_deviceptr ((void *) c),
		 (float *) acc_deviceptr ((void *) b), s);
/* kernel call statement*/
      ipmacc_prompt ((char *) "IPMACC: memory copyout c\n");
      acc_copyout_and_keep ((void *) c, SIZE * SIZE * sizeof (float));
      if (getenv ("IPMACC_VERBOSE"))
	printf ("IPMACC: Synchronizing the region with host\n");
      hipDeviceSynchronize ();



    }
  }
  ipmacc_prompt ((char *) "IPMACC: memory copyout c\n");
  acc_copyout_and_keep ((void *) c, SIZE * SIZE * sizeof (float));



  acc_free (a);
  acc_free (b);
  acc_free (c);
  finish = (float) clock () / (CLOCKS_PER_SEC * 1000);
  elapsed = finish - start;
  fprintf (fil, "%.6lf,", elapsed);
}


void
print ()
{
  int i, j;
  for (i = 0; i < SIZE; ++i)
    {
      for (j = 0; j < SIZE; ++j)
	{
	  fprintf (out, "%f ", c[i * SIZE + j]);
	}
      fprintf (out, "\n");
    }
}

int
main (int argc, char *argv[])
{
  int i;
  int points, var;

  if (argc == 1)
    {
      return;
    }
  points = atoi (argv[1]);
  var = SIZE / points;

  fil = fopen ("time_gpu.csv", "w+");
  out = fopen ("result_gpu.txt", "w+");

  fprintf (fil, "SIZE,matrix sum gpu,\n");


  for (i = (var - 1); i < SIZE; i += var)
    {
      init (i);
      fprintf (fil, "%d,", i);
      sum_GPU (i);
      print ();
      fprintf (fil, "\n");
    }
  fclose (fil);
  fclose (out);
  return 0;
}



__global__ void
__generated_kernel_region_0 (float *a, float *c, float *b, int s)
{
  int __kernel_getuid = threadIdx.x + blockIdx.x * blockDim.x;
  int i;
  int j;
  {
    {


      {
	{


	  {
	    i = 0 + (__kernel_getuid);
	    if (i < s)
	      {
		for (j = 0; j < s; ++j)
		  {
		    c[i * s + j] = a[i * s + j] + b[i * s + j];
		  }
	      }

	  }
	}
      }
    }
  }
}
