#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <openacc.h>
#define IPMACC_MAX1(A)   (A)
#define IPMACC_MAX2(A,B) (A>B?A:B)
#define IPMACC_MAX3(A,B,C) (A>B?(A>C?A:(B>C?B:C)):(B>C?C:B))
#include <hip/hip_runtime.h>



#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <limits.h>
#include <string.h>
#include <openacc.h>
#include <time.h>

#define SIZE 500000

char frase[SIZE];
char palavra[SIZE];

FILE *fil;
FILE *out;


int
init (int s, int p)
{
  int i;
  for (i = 0; i < s; i++)
    {
      frase[i] = 'a';
    }
  frase[i] = '\0';
  for (i = 0; i < p; i++)
    {
      palavra[i] = 'a';
    }
  palavra[i] = '\0';
}




__global__ void __generated_kernel_region_0 (char *frase, int *vector,
					     int parallel_size, int diff,
					     char *palavra, int size_2);

int
string_matching_GPU (int size_1, int size_2)
{
  int i, diff, j, parallel_size, count = 0;
  diff = size_1 - size_2;

  float start, finish, elapsed;
  start = (float) clock () / (CLOCKS_PER_SEC * 1000);
  parallel_size = 100000;
  int vector[parallel_size];
  for (i = 0; i < parallel_size; i++)
    {
      vector[i] = 0;
    }


  ipmacc_prompt ((char *) "IPMACC: memory allocation vector\n");
  acc_create ((void *) vector, parallel_size * sizeof (int));
  ipmacc_prompt ((char *) "IPMACC: memory allocation frase\n");
  acc_create ((void *) frase, SIZE * sizeof (char));
  ipmacc_prompt ((char *) "IPMACC: memory allocation palavra\n");
  acc_create ((void *) palavra, SIZE * sizeof (char));
  ipmacc_prompt ((char *) "IPMACC: memory copyin vector\n");
  acc_copyin ((void *) vector, parallel_size * sizeof (int));
  ipmacc_prompt ((char *) "IPMACC: memory copyin frase\n");
  acc_copyin ((void *) frase, SIZE * sizeof (char));
  ipmacc_prompt ((char *) "IPMACC: memory copyin palavra\n");
  acc_copyin ((void *) palavra, SIZE * sizeof (char));


  {


    {



/* kernel call statement [0, -1]*/
      if (getenv ("IPMACC_VERBOSE"))
	printf ("IPMACC: Launching kernel 0 > gridDim: %d\tblockDim: %d\n",
		(((abs (((diff)) - 0)) / (1))) / 256 + 1, 256);
      __generated_kernel_region_0 <<< (((abs (((diff)) - 0)) / (1))) / 256 +
	1, 256 >>> ((char *) acc_deviceptr ((void *) frase),
		    (int *) acc_deviceptr ((void *) vector), parallel_size,
		    diff, (char *) acc_deviceptr ((void *) palavra), size_2);
/* kernel call statement*/
      if (getenv ("IPMACC_VERBOSE"))
	printf ("IPMACC: Synchronizing the region with host\n");
      hipDeviceSynchronize ();



    }
  }
  ipmacc_prompt ((char *) "IPMACC: memory copyout vector\n");
  acc_copyout_and_keep ((void *) vector, parallel_size * sizeof (int));



  acc_free (frase);
  acc_free (palavra);
  acc_free (vector);
  for (i = 0; i < parallel_size; i++)
    {
      count += vector[i];
    }
  finish = (float) clock () / (CLOCKS_PER_SEC * 1000);
  elapsed = finish - start;
  fprintf (fil, "%.6lf,", elapsed);
  return count;
}

int
main (int argc, char *argv[])
{
  int i, j, k;
  int points, var;
  if (argc == 1)
    {
      return;
    }
  points = atoi (argv[1]);
  var = SIZE / points;

  fil = fopen ("time_gpu.csv", "w+");
  out = fopen ("result_gpu.txt", "w+");

  fprintf (fil, "Size string,Size substring,String Matching GPU,\n");

  for (i = (var - 1); i < SIZE; i += var)
    {
      j = i / 2;
      fprintf (fil, "%d,", i);
      fprintf (fil, "%d,", j);
      init (i, j);
      k = string_matching_GPU (i, j);
      fprintf (out, "%d\n", k);
      fprintf (fil, "\n");
    }
  fclose (fil);
  fclose (out);
  return 0;
}



__global__ void
__generated_kernel_region_0 (char *frase, int *vector, int parallel_size,
			     int diff, char *palavra, int size_2)
{
  int __kernel_getuid = threadIdx.x + blockIdx.x * blockDim.x;
  int i;
  int j;
  {
    {


      {
	{


	  {
	    i = 0 + (__kernel_getuid);
	    if (i < diff)
	      {
		int v;
		v = 0;
		for (j = 0; j < size_2; j++)
		  {
		    if (frase[(i + j)] != palavra[j])
		      {
			v = 1;
		      }
		  }
		if (v == 0)
		  {
		    vector[i % parallel_size]++;
		  }
	      }

	  }
	}
      }
    }
  }
}
