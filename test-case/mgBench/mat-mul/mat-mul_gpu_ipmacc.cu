#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <openacc.h>
#define IPMACC_MAX1(A)   (A)
#define IPMACC_MAX2(A,B) (A>B?A:B)
#define IPMACC_MAX3(A,B,C) (A>B?(A>C?A:(B>C?B:C)):(B>C?C:B))
#include <hip/hip_runtime.h>



#include <stdio.h>
#include <openacc.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

int SIZE;




float *a;
float *b;
float *c;

FILE *fil;
FILE *out;


void
init (int s)
{
  int i, j;
  for (i = 0; i < s; ++i)
    {
      for (j = 0; j < s; ++j)
	{
	  a[i * s + j] = (float) i + j % 100;
	  b[i * s + j] = (float) i + j % 100;
	  c[i * s + j] = 0.0f;
	}
    }
}


void
print (int s)
{
  int i, j;
  for (i = 0; i < s; ++i)
    {
      for (j = 0; j < s; ++j)
	{
	  fprintf (out, "%f ", c[i * s + j]);
	}
      fprintf (out, "\n");
    }
}



__global__ void __generated_kernel_region_0 (float *a, float *c, float *b,
					     int s, float sum);

void
mul_GPU (int s)
{
  int i, j, k, l;
  l = s * s;
  float sum = 0.0;
  float start, finish, elapsed;
  start = (float) clock () / (CLOCKS_PER_SEC * 1000);
  a = (float *) malloc (sizeof (float) * SIZE * SIZE);
  b = (float *) malloc (sizeof (float) * SIZE * SIZE);
  c = (float *) malloc (sizeof (float) * SIZE * SIZE);
  init (SIZE);


  ipmacc_prompt ((char *) "IPMACC: memory allocation c\n");
  acc_create ((void *) c, (l + 0) * sizeof (float));
  ipmacc_prompt ((char *) "IPMACC: memory allocation a\n");
  acc_create ((void *) a, (l + 0) * sizeof (float));
  ipmacc_prompt ((char *) "IPMACC: memory allocation b\n");
  acc_create ((void *) b, (l + 0) * sizeof (float));
  ipmacc_prompt ((char *) "IPMACC: memory copyin c\n");
  acc_copyin ((void *) c, (l + 0) * sizeof (float));
  ipmacc_prompt ((char *) "IPMACC: memory copyin a\n");
  acc_copyin ((void *) a, (l + 0) * sizeof (float));
  ipmacc_prompt ((char *) "IPMACC: memory copyin b\n");
  acc_copyin ((void *) b, (l + 0) * sizeof (float));


  {


    {



/* kernel call statement [0, -1]*/
      if (getenv ("IPMACC_VERBOSE"))
	printf ("IPMACC: Launching kernel 0 > gridDim: %d\tblockDim: %d\n",
		(((abs (((s)) - 0)) / (1))) / 256 + 1, 256);
      __generated_kernel_region_0 <<< (((abs (((s)) - 0)) / (1))) / 256 + 1,
	256 >>> ((float *) acc_deviceptr ((void *) a),
		 (float *) acc_deviceptr ((void *) c),
		 (float *) acc_deviceptr ((void *) b), s, sum);
/* kernel call statement*/
      if (getenv ("IPMACC_VERBOSE"))
	printf ("IPMACC: Synchronizing the region with host\n");
      hipDeviceSynchronize ();



    }
  }
  ipmacc_prompt ((char *) "IPMACC: memory copyout c\n");
  acc_copyout_and_keep ((void *) c, (l + 0) * sizeof (float));



  acc_free (a);
  acc_free (b);
  acc_free (c);
  finish = (float) clock () / (CLOCKS_PER_SEC * 1000);
  elapsed = finish - start;
  fprintf (fil, "%.10lf,", elapsed);
  print (s);
  free (a);
  free (b);
  free (c);
}

int
main (int argc, char *argv[])
{
  int i;
  int points, var, limit;

  if (argc != 2)
    {
      return;
    }
  SIZE = atoi (argv[1]);




  fil = fopen ("time_gpu.csv", "a");
  out = fopen ("result_gpu.txt", "a");

  fprintf (fil, "SIZE,matrix multiplication GPU,\n");



  {

    printf ("i: %d\n", SIZE);




    fprintf (fil, "%d,", SIZE);
    mul_GPU (SIZE);

    fprintf (fil, "\n");



  }
  fclose (fil);
  fclose (out);
  return 0;
}



__global__ void
__generated_kernel_region_0 (float *a, float *c, float *b, int s, float sum)
{
  int __kernel_getuid = threadIdx.x + blockIdx.x * blockDim.x;
  int i;
  int k;
  int j;
  {
    {


      {
	{


	  {
	    i = 0 + (__kernel_getuid);
	    if (i < s)
	      {
		for (j = 0; j < s; ++j)
		  {
		    sum = 0.0;
		    for (k = 0; k < s; ++k)
		      {
			sum = sum + a[i * s + k] * b[k * s + j];
		      }
		    c[i * s + j] = sum;
		  }
	      }

	  }
	}
      }
    }
  }
}
