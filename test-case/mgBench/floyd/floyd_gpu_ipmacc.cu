#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <openacc.h>
#define IPMACC_MAX1(A)   (A)
#define IPMACC_MAX2(A,B) (A>B?A:B)
#define IPMACC_MAX3(A,B,C) (A>B?(A>C?A:(B>C?B:C)):(B>C?C:B))
#include <hip/hip_runtime.h>



#include <stdio.h>
#include <stdlib.h>
#include <openacc.h>
#include <math.h>
#include <limits.h>
#include <time.h>

#define SIZE 1510

int matrix[SIZE * SIZE];
int matrix_dist[SIZE * SIZE];

FILE *fil;
FILE *out;


void
init (int s)
{
  int i, j, r, m;
  for (i = 0; i < s; i++)
    {
      for (j = 0; j < s; j++)
	{
	  matrix[i * s + j] = 99999999;
	  matrix_dist[i * s + j] = 99999999;
	}
    }
  for (i = 0; i < s; i++)
    {
      r = (i * 97) % s;
      for (j = 0; j < r; j++)
	{
	  m = (((j * 1021) * 71 % (s * s)) + 1);
	  matrix[i * s + j] = m;
	}
    }
}



__global__ void __generated_kernel_region_0 (int i, int k, int s,
					     int *matrix_dist);

void
Knearest_GPU (int s)
{
  int i, j, k;
  for (i = 0; i < s; i++)
    {
      for (j = 0; j < s; j++)
	{
	  if (matrix[i * s + j] != 99999999)
	    {
	      matrix_dist[i * s + j] = matrix[i * s + j];
	    }
	}
      matrix_dist[i * s + i] = 0;
    }

  float start, finish, elapsed;
  start = (float) clock () / (CLOCKS_PER_SEC * 1000);


  ipmacc_prompt ((char *) "IPMACC: memory allocation matrix_dist\n");
  acc_create ((void *) matrix_dist, SIZE * SIZE * sizeof (int));
  ipmacc_prompt ((char *) "IPMACC: memory copyin matrix_dist\n");
  acc_copyin ((void *) matrix_dist, SIZE * SIZE * sizeof (int));


  {


    {

      for (k = 0; k < s; k++)
	{

	  for (i = 0; i < s; i++)
	    {



/* kernel call statement [0, -1]*/
	      if (getenv ("IPMACC_VERBOSE"))
		printf
		  ("IPMACC: Launching kernel 0 > gridDim: %d\tblockDim: %d\n",
		   (((abs (((s)) - 0)) / (1))) / 256 + 1, 256);
	      __generated_kernel_region_0 <<< (((abs (((s)) - 0)) / (1))) /
		256 + 1, 256 >>> (i, k, s,
				  (int *) acc_deviceptr ((void *)
							 matrix_dist));
/* kernel call statement*/
	      if (getenv ("IPMACC_VERBOSE"))
		printf ("IPMACC: Synchronizing the region with host\n");
	      hipDeviceSynchronize ();



	    }


	}


    }
  }
  ipmacc_prompt ((char *) "IPMACC: memory copyout matrix_dist\n");
  acc_copyout_and_keep ((void *) matrix_dist, SIZE * SIZE * sizeof (int));



  acc_free (matrix_dist);
  finish = (float) clock () / (CLOCKS_PER_SEC * 1000);
  elapsed = finish - start;
  fprintf (fil, "%.6lf,", elapsed);
}


void
print_distances (int s)
{
  int i, j;
  for (i = 0; i < s; i++)
    {
      for (j = 0; j < s; j++)
	{
	  if (matrix_dist[i * s + j] == 99999999)
	    {
	      continue;
	    }
	  fprintf (out, "%d ", matrix_dist[i * s + j]);
	}
      fprintf (out, "\n");
    }
}

int
main (int argc, char *argv[])
{
  int i;
  int points, var;
  points = atoi (argv[1]);
  var = SIZE / points;

  fil = fopen ("time_gpu.csv", "w+");
  out = fopen ("result_gpu.txt", "w+");

  fprintf (fil, "SIZE,K-nearest GPU\n");
  for (i = (var - 1); i < SIZE; i += var)
    {
      init (i);
      fprintf (fil, "%d,", i);
      Knearest_GPU (i);
      print_distances (i);
      fprintf (fil, "\n");
    }

  fclose (fil);
  fclose (out);
  return 0;
}



__global__ void
__generated_kernel_region_0 (int i, int k, int s, int *matrix_dist)
{
  int __kernel_getuid = threadIdx.x + blockIdx.x * blockDim.x;
  int j;
  {
    {


      {
	{


	  {
	    j = 0 + (__kernel_getuid);
	    if (j < s)
	      {
		if (matrix_dist[i * s + k] != 99999999 &&
		    matrix_dist[k * s + j] != 99999999 &&
		    matrix_dist[i * s + j] >
		    matrix_dist[i * s + k] + matrix_dist[k * s + j])
		  {
		    matrix_dist[i * s + j] =
		      matrix_dist[i * s + k] + matrix_dist[k * s + j];
		  }
	      }

	  }
	}
      }
    }
  }
}
