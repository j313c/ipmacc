#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <openacc.h>
#define IPMACC_MAX1(A)   (A)
#define IPMACC_MAX2(A,B) (A>B?A:B)
#define IPMACC_MAX3(A,B,C) (A>B?(A>C?A:(B>C?B:C)):(B>C?C:B))
#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

typedef struct point
{
  int x;
  int y;
} point;

typedef struct sel_points
{
  int position;
  float value;
} sel_points;

#define SIZE 510
#define default_v 100000.00

point pivots[SIZE];
point the_points[SIZE];
sel_points selected[SIZE * SIZE];

FILE *fil;
FILE *out;

void
init (int s, int t)
{
  int i, j;
  for (i = 0; i < t; i++)
    {
      pivots[i].x = i * 3;
      pivots[i].y = i * 2;
    }
  for (i = 0; i < s; i++)
    {
      the_points[i].x = i * 3;
      the_points[i].y = i * 2;

      for (j = 0; j < s; j++)
	{
	  selected[i * s + j].position = 0;
	  selected[i * s + j].value = default_v;
	}
    }
}

__global__ void __generated_kernel_region_0 (sel_points * selected,
					     point * pivots, int s, int t,
					     point * the_points);

__global__ void __generated_kernel_region_1 (sel_points * selected, int s,
					     int t);

void
k_nearest_gpu (int s, int t)
{
  int i, j, m, q;
  q = s * s;


  ipmacc_prompt ((char *) "IPMACC: memory allocation selected\n");
  acc_create ((void *) selected, SIZE * SIZE * sizeof (sel_points));
  ipmacc_prompt ((char *) "IPMACC: memory allocation pivots\n");
  acc_create ((void *) pivots, SIZE * sizeof (point));
  ipmacc_prompt ((char *) "IPMACC: memory allocation the_points\n");
  acc_create ((void *) the_points, SIZE * sizeof (point));
  ipmacc_prompt ((char *) "IPMACC: memory copyin selected\n");
  acc_copyin ((void *) selected, SIZE * SIZE * sizeof (sel_points));
  ipmacc_prompt ((char *) "IPMACC: memory copyin pivots\n");
  acc_copyin ((void *) pivots, SIZE * sizeof (point));
  ipmacc_prompt ((char *) "IPMACC: memory copyin the_points\n");
  acc_copyin ((void *) the_points, SIZE * sizeof (point));


  {


    {



/* kernel call statement [0, -1]*/
      if (getenv ("IPMACC_VERBOSE"))
	printf ("IPMACC: Launching kernel 0 > gridDim: %d\tblockDim: %d\n",
		(((abs (((t)) - 0)) / (1))) / 256 + 1, 256);
      __generated_kernel_region_0 <<< (((abs (((t)) - 0)) / (1))) / 256 + 1,
	256 >>> ((sel_points *) acc_deviceptr ((void *) selected),
		 (point *) acc_deviceptr ((void *) pivots), s, t,
		 (point *) acc_deviceptr ((void *) the_points));
/* kernel call statement*/
      if (getenv ("IPMACC_VERBOSE"))
	printf ("IPMACC: Synchronizing the region with host\n");
      hipDeviceSynchronize ();







/* kernel call statement [0, -1]*/
      if (getenv ("IPMACC_VERBOSE"))
	printf ("IPMACC: Launching kernel 1 > gridDim: %d\tblockDim: %d\n",
		(((abs (((t)) - 0)) / (1))) / 256 + 1, 256);
      __generated_kernel_region_1 <<< (((abs (((t)) - 0)) / (1))) / 256 + 1,
	256 >>> ((sel_points *) acc_deviceptr ((void *) selected), s, t);
/* kernel call statement*/
      if (getenv ("IPMACC_VERBOSE"))
	printf ("IPMACC: Synchronizing the region with host\n");
      hipDeviceSynchronize ();



    }
  }
  ipmacc_prompt ((char *) "IPMACC: memory copyout selected\n");
  acc_copyout_and_keep ((void *) selected, SIZE * SIZE * sizeof (sel_points));



  acc_free (pivots);
  acc_free (the_points);
  acc_free (selected);
}

void
print (int s, int k, int t)
{
  int i;
  for (i = 0; i < t; i++)
    {
      int j;
      for (j = 0; j <= k; j++)
	{
	  int pos_sel;
	  pos_sel = selected[i * s + j].position;
	  fprintf (out,
		   "pivot position %d (%d ; %d), point nearest %d (%d ; %d), distance %.0f\n",
		   i, pivots[i].x, pivots[i].y, pos_sel,
		   the_points[pos_sel].x, the_points[pos_sel].y,
		   selected[i * s + j].value);
	}
      fprintf (out, "\n");
    }
  fprintf (out, "\n");
}

int
main (int argc, char *argv[])
{
  fil = fopen ("time_gpu.csv", "w+");
  out = fopen ("result_gpu.txt", "w+");

  int i, j, points, var;
  if (argc < 2)
    {
      printf ("Number of points is unknown.\n");
      return;
    }
  points = atoi (argv[1]);
  var = SIZE / points;


  fprintf (fil, "SIZE, K, K-nearest gpu time,\n");
  for (i = (var - 1); i < SIZE; i += var)
    {
      j = i / 2;
      fprintf (fil, "%d,%d,", i, j);
      init (i, j);



      float start, finish, elapsed;
      start = (float) clock () / (CLOCKS_PER_SEC * 1000);

      k_nearest_gpu (i, j);

      finish = (float) clock () / (CLOCKS_PER_SEC * 1000);
      elapsed = finish - start;
      fprintf (fil, "%.9lf,", elapsed);


      print (i, j, j);
      fprintf (fil, "\n");
    }

  fclose (fil);
  fclose (out);
  return 0;
}



__global__ void
__generated_kernel_region_0 (sel_points * selected, point * pivots, int s,
			     int t, point * the_points)
{
  int __kernel_getuid = threadIdx.x + blockIdx.x * blockDim.x;
  int i;
  int j;
  {
    {


      {
	{


	  {
	    i = 0 + (__kernel_getuid);
	    if (i < t)
	      {
		for (j = 0; j < s; j++)
		  {
		    float distance, x, y;
		    x = pivots[i].x - the_points[j].x;
		    y = pivots[i].y - the_points[j].y;
		    x = x * x;
		    y = y * y;

		    distance = x + y;
		    distance = sqrt (distance);

		    selected[i * s + j].value = distance;
		    selected[i * s + j].position = j;
		  }
	      }

	  }
	}
      }
    }
  }
}

__global__ void
__generated_kernel_region_1 (sel_points * selected, int s, int t)
{
  int __kernel_getuid = threadIdx.x + blockIdx.x * blockDim.x;
  int i;
  int j;
  int m;
  {
    {


      {
	{


	  {
	    i = 0 + (__kernel_getuid);
	    if (i < t)
	      {


		int j;
		for (j = 0; j < s; j++)
		  {
		    int m;
		    for (m = j + 1; m < s; m++)
		      {
			if (selected[i * s + j].value >
			    selected[i * s + m].value)
			  {
			    sel_points aux;
			    aux = selected[i * s + j];
			    selected[i * s + j] = selected[i * s + m];
			    selected[i * s + m] = aux;
			  }
		      }
		  }
	      }

	  }
	}
      }
    }
  }
}
